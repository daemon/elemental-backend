
#include <hip/hip_runtime.h>
#include <iostream>

// Only works with 3x3x3 filters for now
__global__ void conv3d_gpu(float *field, float *filter, float *result, int fieldLength, int filterLength) {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  int fieldVolume = fieldLength * fieldLength * fieldLength;
  int fieldArea = fieldLength * fieldLength;
  if (gid >= fieldVolume)
    return;
  int base = -fieldArea - fieldLength - 1;
  int fieldIndex = 0;
  double answer = 0.0;
  for (int i = 0; i < filterLength; ++i) {
    for (int j = 0; j < filterLength; ++j) {
      int boundary = (gid + base + 1) / fieldLength;
      for (int k = 0; k < filterLength; ++k) {
        fieldIndex = gid + base + k;
        if (fieldIndex / fieldLength != boundary)
          continue;
        if (fieldIndex < 0 || fieldIndex >= fieldVolume)
          continue;
        answer += filter[filterLength * filterLength * i + filterLength * j + k] * field[fieldIndex];
      }
      base += fieldLength;
    }
    base -= filterLength * fieldLength;
    base += fieldArea;
  }
  result[gid] = answer;
}

__global__ void heat3d_gpu(float *field, float *laplacian, float *alpha, float dt) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  field[gid] += laplacian[gid] * alpha[gid] * dt;
}

// Simple trilinear interpolation
__global__ void advection_gpu(float *fieldU, float *fieldV, float *fieldW, float *resultU, float *resultV, float *resultW, int fieldLength, float dt) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int volume = fieldLength * fieldLength * fieldLength;
  int area = fieldLength * fieldLength;
  if (gid >= volume)
    return;
  int u = gid / area;
  int v = (gid - u * area) / fieldLength;
  int w = (gid - u * area) % fieldLength;
  float u2 = u - dt * fieldU[gid];
  float v2 = v - dt * fieldV[gid];
  float w2 = w - dt * fieldW[gid];
  int c1 = (int) u2;
  int c2 = c1 + 1;
  int c3 = (int) v2;
  int c4 = c3 + 1;
  int c5 = (int) w2;
  int c6 = c5 + 1;
  float pctU = (u2 - c1);
  float pctV = (v2 - c3);
  float pctW = (w2 - c5);
  int i1 = c2 * area + c3 * fieldLength + c5;
  int i2 = c1 * area + c3 * fieldLength + c5;
  int i3 = c2 * area + c4 * fieldLength + c5;
  int i4 = c1 * area + c4 * fieldLength + c5;
  int i5 = c2 * area + c3 * fieldLength + c6;
  int i6 = c1 * area + c3 * fieldLength + c6;
  int i7 = c2 * area + c4 * fieldLength + c6;
  int i8 = c1 * area + c4 * fieldLength + c6;
  float uI1 = 0 ? c2 >= fieldLength || c3 < 0 || c5 < 0 : fieldU[i1];
  float uI2 = 0 ? c1 < 0 || c3 < 0 || c5 < 0 : fieldU[i2];
  float uI6 = 0 ? c1 < 0 || c3 < 0 || c6 >= fieldLength : fieldU[i6];
  float uI5 = 0 ? c2 >= fieldLength || c3 < 0 || c6 >= fieldLength : fieldU[i5];
  float uI4 = 0 ? c1 < 0 || c4 >= fieldLength || c5 < 0 : fieldU[i4];
  float uI3 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c5 < 0 : fieldU[i3];
  float uI8 = 0 ? c1 < 0 || c4 >= fieldLength || c6 >= fieldLength : fieldU[i8];
  float uI7 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c6 >= fieldLength : fieldU[i7];
  float top1, top2, top, bot1, bot2, bot;
  top1 = pctU * uI1 + (1 - pctU) * uI2;
  top2 = pctU * uI6 + (1 - pctU) * uI5;
  top = pctW * top2 + (1 - pctW) * top1;
  bot1 = pctU * uI4 + (1 - pctU) * uI3;
  bot2 = pctU * uI8 + (1 - pctU) * uI7;
  bot = pctW * bot2 + (1 - pctW) * bot1;
  float rU = pctV * bot + (1 - pctV) * top;
  float vI1 = 0 ? c2 >= fieldLength || c3 < 0 || c5 < 0 : fieldV[i1];
  float vI2 = 0 ? c1 < 0 || c3 < 0 || c5 < 0 : fieldV[i2];
  float vI6 = 0 ? c1 < 0 || c3 < 0 || c6 >= fieldLength : fieldV[i6];
  float vI5 = 0 ? c2 >= fieldLength || c3 < 0 || c6 >= fieldLength : fieldV[i5];
  float vI4 = 0 ? c1 < 0 || c4 >= fieldLength || c5 < 0 : fieldV[i4];
  float vI3 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c5 < 0 : fieldV[i3];
  float vI8 = 0 ? c1 < 0 || c4 >= fieldLength || c6 >= fieldLength : fieldV[i8];
  float vI7 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c6 >= fieldLength : fieldV[i7];
  top1 = pctU * vI1 + (1 - pctU) * vI2;
  top2 = pctU * vI6 + (1 - pctU) * vI5;
  top = pctW * top2 + (1 - pctW) * top1;
  bot1 = pctU * vI4 + (1 - pctU) * vI3;
  bot2 = pctU * vI8 + (1 - pctU) * vI7;
  bot = pctW * bot2 + (1 - pctW) * bot1;
  float rV = pctV * bot + (1 - pctV) * top;
  float wI1 = 0 ? c2 >= fieldLength || c3 < 0 || c5 < 0 : fieldW[i1];
  float wI2 = 0 ? c1 < 0 || c3 < 0 || c5 < 0 : fieldW[i2];
  float wI6 = 0 ? c1 < 0 || c3 < 0 || c6 >= fieldLength : fieldW[i6];
  float wI5 = 0 ? c2 >= fieldLength || c3 < 0 || c6 >= fieldLength : fieldW[i5];
  float wI4 = 0 ? c1 < 0 || c4 >= fieldLength || c5 < 0 : fieldW[i4];
  float wI3 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c5 < 0 : fieldW[i3];
  float wI8 = 0 ? c1 < 0 || c4 >= fieldLength || c6 >= fieldLength : fieldW[i8];
  float wI7 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c6 >= fieldLength : fieldW[i7];
  top1 = pctU * wI1 + (1 - pctU) * wI2;
  top2 = pctU * wI6 + (1 - pctU) * wI5;
  top = pctW * top2 + (1 - pctW) * top1;
  bot1 = pctU * wI4 + (1 - pctU) * wI3;
  bot2 = pctU * wI8 + (1 - pctU) * wI7;
  bot = pctW * bot2 + (1 - pctW) * bot1;
  float rW = pctV * bot + (1 - pctV) * top;
  resultU[gid] = rU;
  resultV[gid] = rV;
  resultW[gid] = rW;
}

extern "C"
__declspec(dllexport) void __cdecl
conv3d(float *field, float *filter, float *result, int fieldLength, int filterLength) {
  float *dField, *dFilter, *dResult;
  size_t fieldSize = sizeof(float) * fieldLength * fieldLength * fieldLength;
  size_t filterSize = sizeof(float) * filterLength * filterLength * filterLength;
  hipMalloc((void **) &dField, fieldSize);
  hipMalloc((void **) &dFilter, filterSize);
  hipMalloc((void **) &dResult, fieldSize);
  hipMemcpy(dField, field, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFilter, filter, filterSize, hipMemcpyHostToDevice);
  int nBlocks = (fieldLength * fieldLength * fieldLength) / 16 + 1;
  conv3d_gpu<<<nBlocks, 16>>>(dField, dFilter, dResult, fieldLength, filterLength);
  hipDeviceSynchronize();
  hipMemcpy(result, dResult, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFilter);
  hipFree(dResult);
  hipFree(dField);
}

extern "C"
__declspec(dllexport) void __cdecl
heat3d(float *field, float *buffer, float *alpha, float dt, int fieldLength) {
  float *dField, *dFilter, *dResult, *dAlpha;
  float filter[27] = {0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, -6, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0};
  size_t fieldSize = sizeof(float) * fieldLength * fieldLength * fieldLength;
  size_t filterSize = sizeof(float) * 27;
  hipMalloc((void **) &dField, fieldSize);
  hipMalloc((void **) &dFilter, filterSize);
  hipMalloc((void **) &dResult, fieldSize);
  hipMemcpy(dField, field, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFilter, filter, filterSize, hipMemcpyHostToDevice);
  int nBlocks = (fieldLength * fieldLength * fieldLength) / 16 + 1;
  conv3d_gpu<<<nBlocks, 16>>>(dField, dFilter, dResult, fieldLength, 3);
  hipMalloc((void **) &dAlpha, fieldSize);
  hipMemcpy(dAlpha, alpha, fieldSize, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  heat3d_gpu<<<nBlocks, 16>>>(dField, dResult, dAlpha, dt);
  hipDeviceSynchronize();
  hipMemcpy(field, dField, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFilter);
  hipFree(dResult);
  hipFree(dField);
  hipFree(dAlpha);
}

extern "C"
__declspec(dllexport) void __cdecl
advection(float *fieldU, float *fieldV, float *fieldW, float dt, int fieldLength) {
  float *dFieldU, *dFieldV, *dFieldW, *dResultU, *dResultV, *dResultW;
  size_t fieldSize = fieldLength * fieldLength * fieldLength * sizeof(float);
  hipMalloc((void **) &dFieldU, fieldSize);
  hipMalloc((void **) &dFieldV, fieldSize);
  hipMalloc((void **) &dFieldW, fieldSize);
  hipMalloc((void **) &dResultU, fieldSize);
  hipMalloc((void **) &dResultV, fieldSize);
  hipMalloc((void **) &dResultW, fieldSize);
  hipMemcpy(dFieldU, fieldU, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFieldV, fieldV, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFieldW, fieldW, fieldSize, hipMemcpyHostToDevice);
  int nBlocks = (fieldLength * fieldLength * fieldLength) / 16 + 1;
  advection_gpu<<<nBlocks, 16>>>(dFieldU, dFieldV, dFieldW, dResultU, dResultV, dResultW, fieldLength, dt);
  hipDeviceSynchronize();
  hipMemcpy(fieldU, dResultU, fieldSize, hipMemcpyDeviceToHost);
  hipMemcpy(fieldV, dResultV, fieldSize, hipMemcpyDeviceToHost);
  hipMemcpy(fieldW, dResultW, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFieldU);
  hipFree(dFieldV);
  hipFree(dFieldW);
  hipFree(dResultU);
  hipFree(dResultV);
  hipFree(dResultW);
}

__global__ void sph_density_gpu(float *posX, float *posY, float *posZ, float *density, int nParticles) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= nParticles)
    return;
  float rho = 0;  
  float x = posX[gid];
  float y = posY[gid];
  float z = posZ[gid];
  float pi = 3.141592654f;
  for (int i = 0; i < nParticles; ++i) {
    float dX = posX[i] - x;
    float dY = posY[i] - y;
    float dZ = posZ[i] - z;
    float dist = dX * dX + dY * dY + dZ * dZ;
    if (dist <= 1)
      rho += (1 - (3.0 / 2) * dist * (1 - sqrtf(dist) / 2)) * (1 / (pi * 8));
    else if (dist <= 4)
      rho += powf(2 - sqrtf(dist), 3) * (1 / (4 * pi * 8));
  }
  density[gid] = rho;
}

__global__ void sph_accel_gpu(float *posX, float *posY, float *posZ, float *velX, float *velY, float *velZ, float *density, bool *wallBlocks, float dt, int nParticles, int fieldLength) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid >= nParticles)
    return;
  float3 a = make_float3(0, -9.81, 0);
  float x = posX[gid];
  float y = posY[gid];
  float z = posZ[gid];
  float pi = 3.141592654f;
  float rhoi = density[gid];
  float3 force = make_float3(0, 0, 0);
  for (int i = 0; i < nParticles; ++i) {
    if (i == gid)
      continue;
    float3 dX = make_float3(x - posX[i], y - posY[i], z - posZ[i]);
    float dist = dX.x * dX.x + dX.y * dX.y + dX.z * dX.z;
    if (dist > 4)
      continue;
    float rhoj = density[i];
    float qij = max(sqrtf(dist) / 2, 0.0005);
    float k1 = 1 / (pi * 16 * rhoj) * (1 - qij) * (70 * (rhoi + rhoj - 0.2) * (1 - qij) / qij);
    force.x += k1 * dX.x;
    force.y += k1 * dX.y;
    force.z += k1 * dX.z;
  }
  a.x = force.x / rhoi;
  a.y = force.y / rhoi - 9.81;
  a.z = force.z / rhoi;
  velX[gid] += a.x * dt;
  velY[gid] += a.y * dt;
  velZ[gid] += a.z * dt;
  int newX = (int) (posX[gid] + velX[gid] * dt);
  int newY = (int) (posY[gid] + velY[gid] * dt);
  int newZ = (int) (posZ[gid] + velZ[gid] * dt);
  if (newX >= fieldLength || newX < 0) {
    velX[gid] = -velX[gid];
    velX[gid] *= 0.15;
  }
  if (newY >= fieldLength || newY < 0) {
    velY[gid] = -velY[gid];
    velY[gid] *= 0.15;
  }
  if (newZ >= fieldLength || newZ < 0) {
    velZ[gid] = -velZ[gid];
    velZ[gid] *= 0.15;
  }
  if (wallBlocks[newX * fieldLength * fieldLength + newY * fieldLength + newZ]) {
    if (wallBlocks[newX * fieldLength * fieldLength + fieldLength * (int) (posY[gid]) + (int) (posZ[gid])]) {
      velX[gid] = -velX[gid];
      velX[gid] *= 0.15;
    }
    if (wallBlocks[(int) (posX[gid]) * fieldLength * fieldLength + fieldLength * newY + (int) (posZ[gid])]) {
      velY[gid] = -velY[gid];
      velY[gid] *= 0.15;
    }
    if (wallBlocks[(int) (posX[gid]) * fieldLength * fieldLength + fieldLength * (int) (posY[gid]) + newZ]) {
      velZ[gid] = -velZ[gid];
      velZ[gid] *= 0.15;
    }
  }
  posX[gid] += velX[gid] * dt;
  posY[gid] += velY[gid] * dt;
  posZ[gid] += velZ[gid] * dt;
}

extern "C"
__declspec(dllexport) void __cdecl
sph(float *posX, float *posY, float *posZ, float *velX, float *velY, float *velZ, bool *wallBlocks, float dt, int nParticles, int fieldLength) {
  float *dPosX,*dPosY, *dPosZ, *dVelX, *dVelY, *dVelZ, *dDensity;
  bool *dWallBlocks;
  int particlesSize = nParticles * sizeof(float);
  hipMalloc((void **) &dPosX, particlesSize);
  hipMalloc((void **) &dPosY, particlesSize);
  hipMalloc((void **) &dPosZ, particlesSize);
  hipMalloc((void **) &dVelX, particlesSize);
  hipMalloc((void **) &dVelY, particlesSize);
  hipMalloc((void **) &dVelZ, particlesSize);
  hipMalloc((void **) &dDensity, particlesSize);
  hipMalloc((void **) &dWallBlocks, fieldLength * fieldLength * fieldLength * sizeof(bool));
  hipMemcpy(dPosX, posX, particlesSize, hipMemcpyHostToDevice);
  hipMemcpy(dPosY, posY, particlesSize, hipMemcpyHostToDevice);
  hipMemcpy(dPosZ, posZ, particlesSize, hipMemcpyHostToDevice);
  hipMemcpy(dVelX, velX, particlesSize, hipMemcpyHostToDevice);
  hipMemcpy(dVelY, velY, particlesSize, hipMemcpyHostToDevice);
  hipMemcpy(dVelZ, velZ, particlesSize, hipMemcpyHostToDevice);
  hipMemcpy(dWallBlocks, wallBlocks, fieldLength * fieldLength * fieldLength * sizeof(bool), hipMemcpyHostToDevice);
  int nBlocks = nParticles / 16 + 1;
  sph_density_gpu<<<nBlocks, 16>>>(dPosX, dPosY, dPosZ, dDensity, nParticles);
  sph_accel_gpu<<<nBlocks, 16>>>(dPosX, dPosY, dPosZ, dVelX, dVelY, dVelZ, dDensity, dWallBlocks, dt, nParticles, fieldLength);
  hipMemcpy(velX, dVelX, particlesSize, hipMemcpyDeviceToHost);
  hipMemcpy(velY, dVelY, particlesSize, hipMemcpyDeviceToHost);
  hipMemcpy(velZ, dVelZ, particlesSize, hipMemcpyDeviceToHost);
  hipMemcpy(posX, dPosX, particlesSize, hipMemcpyDeviceToHost);
  hipMemcpy(posY, dPosY, particlesSize, hipMemcpyDeviceToHost);
  hipMemcpy(posZ, dPosZ, particlesSize, hipMemcpyDeviceToHost);
  hipFree(dPosX);
  hipFree(dPosY);
  hipFree(dPosZ);
  hipFree(dVelX);
  hipFree(dVelY);
  hipFree(dVelZ);
  hipFree(dDensity);
  hipFree(dWallBlocks);
}
