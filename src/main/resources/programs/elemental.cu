
#include <hip/hip_runtime.h>
#include <iostream>

// Only works with 3x3x3 filters for now
__global__ void conv3d_gpu(float *field, float *filter, float *result, int fieldLength, int filterLength) {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  int fieldVolume = fieldLength * fieldLength * fieldLength;
  int fieldArea = fieldLength * fieldLength;
  if (gid >= fieldVolume)
    return;
  int base = -fieldArea - fieldLength - 1;
  int fieldIndex = 0;
  double answer = 0.0;
  for (int i = 0; i < filterLength; ++i) {
    for (int j = 0; j < filterLength; ++j) {
      int boundary = (gid + base + 1) / fieldLength;
      for (int k = 0; k < filterLength; ++k) {
        fieldIndex = gid + base + k;
        if (fieldIndex / fieldLength != boundary)
          continue;
        if (fieldIndex < 0 || fieldIndex >= fieldVolume)
          continue;
        answer += filter[filterLength * filterLength * i + filterLength * j + k] * field[fieldIndex];
      }
      base += fieldLength;
    }
    base -= filterLength * fieldLength;
    base += fieldArea;
  }
  result[gid] = answer;
}

__global__ void heat3d_gpu(float *field, float *laplacian, float *alpha, float dt) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  field[gid] += laplacian[gid] * alpha[gid] * dt;
}

// Simple trilinear interpolation
__global__ void advection_gpu(float *fieldU, float *fieldV, float *fieldW, float *resultU, float *resultV, float *resultW, int fieldLength, float dt) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  int volume = fieldLength * fieldLength * fieldLength;
  int area = fieldLength * fieldLength;
  if (gid >= volume)
    return;
  int u = gid / area;
  int v = (gid - u * area) / fieldLength;
  int w = (gid - u * area) % fieldLength;
  float u2 = u - dt * fieldU[gid];
  float v2 = v - dt * fieldV[gid];
  float w2 = w - dt * fieldW[gid];
  int c1 = (int) u2;
  int c2 = c1 + 1;
  int c3 = (int) v2;
  int c4 = c3 + 1;
  int c5 = (int) w2;
  int c6 = c5 + 1;
  float pctU = (u2 - c1);
  float pctV = (v2 - c3);
  float pctW = (w2 - c5);
  int i1 = c2 * area + c3 * fieldLength + c5;
  int i2 = c1 * area + c3 * fieldLength + c5;
  int i3 = c2 * area + c4 * fieldLength + c5;
  int i4 = c1 * area + c4 * fieldLength + c5;
  int i5 = c2 * area + c3 * fieldLength + c6;
  int i6 = c1 * area + c3 * fieldLength + c6;
  int i7 = c2 * area + c4 * fieldLength + c6;
  int i8 = c1 * area + c4 * fieldLength + c6;
  float uI1 = 0 ? c2 >= fieldLength || c3 < 0 || c5 < 0 : fieldU[i1];
  float uI2 = 0 ? c1 < 0 || c3 < 0 || c5 < 0 : fieldU[i2];
  float uI6 = 0 ? c1 < 0 || c3 < 0 || c6 >= fieldLength : fieldU[i6];
  float uI5 = 0 ? c2 >= fieldLength || c3 < 0 || c6 >= fieldLength : fieldU[i5];
  float uI4 = 0 ? c1 < 0 || c4 >= fieldLength || c5 < 0 : fieldU[i4];
  float uI3 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c5 < 0 : fieldU[i3];
  float uI8 = 0 ? c1 < 0 || c4 >= fieldLength || c6 >= fieldLength : fieldU[i8];
  float uI7 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c6 >= fieldLength : fieldU[i7];
  float top1, top2, top, bot1, bot2, bot;
  top1 = pctU * uI1 + (1 - pctU) * uI2;
  top2 = pctU * uI6 + (1 - pctU) * uI5;
  top = pctW * top2 + (1 - pctW) * top1;
  bot1 = pctU * uI4 + (1 - pctU) * uI3;
  bot2 = pctU * uI8 + (1 - pctU) * uI7;
  bot = pctW * bot2 + (1 - pctW) * bot1;
  float rU = pctV * bot + (1 - pctV) * top;
  float vI1 = 0 ? c2 >= fieldLength || c3 < 0 || c5 < 0 : fieldV[i1];
  float vI2 = 0 ? c1 < 0 || c3 < 0 || c5 < 0 : fieldV[i2];
  float vI6 = 0 ? c1 < 0 || c3 < 0 || c6 >= fieldLength : fieldV[i6];
  float vI5 = 0 ? c2 >= fieldLength || c3 < 0 || c6 >= fieldLength : fieldV[i5];
  float vI4 = 0 ? c1 < 0 || c4 >= fieldLength || c5 < 0 : fieldV[i4];
  float vI3 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c5 < 0 : fieldV[i3];
  float vI8 = 0 ? c1 < 0 || c4 >= fieldLength || c6 >= fieldLength : fieldV[i8];
  float vI7 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c6 >= fieldLength : fieldV[i7];
  top1 = pctU * vI1 + (1 - pctU) * vI2;
  top2 = pctU * vI6 + (1 - pctU) * vI5;
  top = pctW * top2 + (1 - pctW) * top1;
  bot1 = pctU * vI4 + (1 - pctU) * vI3;
  bot2 = pctU * vI8 + (1 - pctU) * vI7;
  bot = pctW * bot2 + (1 - pctW) * bot1;
  float rV = pctV * bot + (1 - pctV) * top;
  float wI1 = 0 ? c2 >= fieldLength || c3 < 0 || c5 < 0 : fieldW[i1];
  float wI2 = 0 ? c1 < 0 || c3 < 0 || c5 < 0 : fieldW[i2];
  float wI6 = 0 ? c1 < 0 || c3 < 0 || c6 >= fieldLength : fieldW[i6];
  float wI5 = 0 ? c2 >= fieldLength || c3 < 0 || c6 >= fieldLength : fieldW[i5];
  float wI4 = 0 ? c1 < 0 || c4 >= fieldLength || c5 < 0 : fieldW[i4];
  float wI3 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c5 < 0 : fieldW[i3];
  float wI8 = 0 ? c1 < 0 || c4 >= fieldLength || c6 >= fieldLength : fieldW[i8];
  float wI7 = 0 ? c2 >= fieldLength || c4 >= fieldLength || c6 >= fieldLength : fieldW[i7];
  top1 = pctU * wI1 + (1 - pctU) * wI2;
  top2 = pctU * wI6 + (1 - pctU) * wI5;
  top = pctW * top2 + (1 - pctW) * top1;
  bot1 = pctU * wI4 + (1 - pctU) * wI3;
  bot2 = pctU * wI8 + (1 - pctU) * wI7;
  bot = pctW * bot2 + (1 - pctW) * bot1;
  float rW = pctV * bot + (1 - pctV) * top;
  resultU[gid] = rU;
  resultV[gid] = rV;
  resultW[gid] = rW;
}

extern "C"
__declspec(dllexport) void __cdecl
conv3d(float *field, float *filter, float *result, int fieldLength, int filterLength) {
  float *dField, *dFilter, *dResult;
  size_t fieldSize = sizeof(float) * fieldLength * fieldLength * fieldLength;
  size_t filterSize = sizeof(float) * filterLength * filterLength * filterLength;
  hipMalloc((void **) &dField, fieldSize);
  hipMalloc((void **) &dFilter, filterSize);
  hipMalloc((void **) &dResult, fieldSize);
  hipMemcpy(dField, field, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFilter, filter, filterSize, hipMemcpyHostToDevice);
  int nBlocks = (fieldLength * fieldLength * fieldLength) / 16 + 1;
  conv3d_gpu<<<nBlocks, 16>>>(dField, dFilter, dResult, fieldLength, filterLength);
  hipDeviceSynchronize();
  hipMemcpy(result, dResult, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFilter);
  hipFree(dResult);
  hipFree(dField);
}

extern "C"
__declspec(dllexport) void __cdecl
heat3d(float *field, float *buffer, float *alpha, float dt, int fieldLength) {
  float *dField, *dFilter, *dResult, *dAlpha;
  float filter[27] = {0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, -6, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0};
  size_t fieldSize = sizeof(float) * fieldLength * fieldLength * fieldLength;
  size_t filterSize = sizeof(float) * 27;
  hipMalloc((void **) &dField, fieldSize);
  hipMalloc((void **) &dFilter, filterSize);
  hipMalloc((void **) &dResult, fieldSize);
  hipMemcpy(dField, field, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFilter, filter, filterSize, hipMemcpyHostToDevice);
  int nBlocks = (fieldLength * fieldLength * fieldLength) / 16 + 1;
  conv3d_gpu<<<nBlocks, 16>>>(dField, dFilter, dResult, fieldLength, 3);
  hipMalloc((void **) &dAlpha, fieldSize);
  hipMemcpy(dAlpha, alpha, fieldSize, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  heat3d_gpu<<<nBlocks, 16>>>(dField, dResult, dAlpha, dt);
  hipDeviceSynchronize();
  hipMemcpy(field, dField, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFilter);
  hipFree(dResult);
  hipFree(dField);
  hipFree(dAlpha);
}

extern "C"
__declspec(dllexport) void __cdecl
advection(float *fieldU, float *fieldV, float *fieldW, float dt, int fieldLength) {
  float *dFieldU, *dFieldV, *dFieldW, *dResultU, *dResultV, *dResultW;
  size_t fieldSize = fieldLength * fieldLength * fieldLength * sizeof(float);
  hipMalloc((void **) &dFieldU, fieldSize);
  hipMalloc((void **) &dFieldV, fieldSize);
  hipMalloc((void **) &dFieldW, fieldSize);
  hipMalloc((void **) &dResultU, fieldSize);
  hipMalloc((void **) &dResultV, fieldSize);
  hipMalloc((void **) &dResultW, fieldSize);
  hipMemcpy(dFieldU, fieldU, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFieldV, fieldV, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFieldW, fieldW, fieldSize, hipMemcpyHostToDevice);
  int nBlocks = (fieldLength * fieldLength * fieldLength) / 16 + 1;
  advection_gpu<<<nBlocks, 16>>>(dFieldU, dFieldV, dFieldW, dResultU, dResultV, dResultW, fieldLength, dt);
  hipDeviceSynchronize();
  hipMemcpy(fieldU, dResultU, fieldSize, hipMemcpyDeviceToHost);
  hipMemcpy(fieldV, dResultV, fieldSize, hipMemcpyDeviceToHost);
  hipMemcpy(fieldW, dResultW, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFieldU);
  hipFree(dFieldV);
  hipFree(dFieldW);
  hipFree(dResultU);
  hipFree(dResultV);
  hipFree(dResultW);
}