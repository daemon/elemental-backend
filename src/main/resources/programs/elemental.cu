
#include <hip/hip_runtime.h>
#include <iostream>

// Only works with 3x3x3 filters for now
__global__ void conv3d_gpu(float *field, float *filter, float *result, int fieldLength, int filterLength) {
  int gid = blockDim.x * blockIdx.x + threadIdx.x;
  int fieldVolume = fieldLength * fieldLength * fieldLength;
  int fieldArea = fieldLength * fieldLength;
  if (gid >= fieldVolume)
    return;
  int base = -fieldArea - fieldLength - 1;
  int fieldIndex = 0;
  double answer = 0.0;
  for (int i = 0; i < filterLength; ++i) {
    for (int j = 0; j < filterLength; ++j) {
      int boundary = (gid + base + 1) / fieldLength;
      for (int k = 0; k < filterLength; ++k) {
        fieldIndex = gid + base + k;
        if (fieldIndex / fieldLength != boundary)
          continue;
        if (fieldIndex < 0 || fieldIndex >= fieldVolume)
          continue;
        answer += filter[filterLength * filterLength * i + filterLength * j + k] * field[fieldIndex];
      }
      base += fieldLength;
    }
    base -= filterLength * fieldLength;
    base += fieldArea;
  }
  result[gid] = answer;
}

__global__ void heat3d_gpu(float *field, float *laplacian, float *alpha, float dt) {
  int gid = blockIdx.x;
  field[gid] += laplacian[gid] * alpha[gid] * dt;
}

extern "C"
__declspec(dllexport) void __cdecl
conv3d(float *field, float *filter, float *result, int fieldLength, int filterLength) {
  float *dField, *dFilter, *dResult;
  size_t fieldSize = sizeof(float) * fieldLength * fieldLength * fieldLength;
  size_t filterSize = sizeof(float) * filterLength * filterLength * filterLength;
  hipMalloc((void **) &dField, fieldSize);
  hipMalloc((void **) &dFilter, filterSize);
  hipMalloc((void **) &dResult, fieldSize);
  hipMemcpy(dField, field, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFilter, filter, filterSize, hipMemcpyHostToDevice);
  int nBlocks = (fieldLength * fieldLength * fieldLength) / 16 + 1;
  conv3d_gpu<<<nBlocks, 16>>>(dField, dFilter, dResult, fieldLength, filterLength);
  hipDeviceSynchronize();
  hipMemcpy(result, dResult, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFilter);
  hipFree(dResult);
  hipFree(dField);
}

extern "C"
__declspec(dllexport) void __cdecl
heat3d(float *field, float *buffer, float *alpha, float dt, int fieldLength) {
  float *dField, *dFilter, *dResult, *dAlpha;
  float filter[27] = {0, 0, 0, 0, 1, 0, 0, 0, 0, 0, 1, 0, 1, -6, 1, 0, 1, 0, 0, 0, 0, 0, 1, 0, 0, 0, 0};
  size_t fieldSize = sizeof(float) * fieldLength * fieldLength * fieldLength;
  size_t filterSize = sizeof(float) * 27;
  hipMalloc((void **) &dField, fieldSize);
  hipMalloc((void **) &dFilter, filterSize);
  hipMalloc((void **) &dResult, fieldSize);
  hipMemcpy(dField, field, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFilter, filter, filterSize, hipMemcpyHostToDevice);
  int nBlocks = (fieldLength * fieldLength * fieldLength) / 16 + 1;
  conv3d_gpu<<<nBlocks, 16>>>(dField, dFilter, dResult, fieldLength, 3);
  hipMalloc((void **) &dAlpha, fieldSize);
  hipMemcpy(dAlpha, alpha, fieldSize, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  heat3d_gpu<<<fieldLength * fieldLength * fieldLength, 1>>>(dField, dResult, dAlpha, dt);
  hipDeviceSynchronize();
  hipMemcpy(field, dField, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFilter);
  hipFree(dResult);
  hipFree(dField);
  hipFree(dAlpha);
}