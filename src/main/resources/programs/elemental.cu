
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void conv3d_gpu(float *field, float *filter, float *result, int fieldLength, int filterLength) {
  int gid = blockIdx.x;
  int base = -fieldLength * fieldLength - fieldLength - 1;
  int fieldIndex = 0;
  double answer = 0.0;
  for (int i = 0; i < filterLength; ++i) {
    for (int j = 0; j < filterLength; ++j) {
      int boundary = (gid + base + 1) / fieldLength;
      for (int k = 0; k < filterLength; ++k) {
        fieldIndex = gid + base + k;
        if (fieldIndex / fieldLength != boundary)
          continue;
        if (fieldIndex < 0 || fieldIndex >= fieldLength * fieldLength * fieldLength)
          continue;
        double fieldValue = field[fieldIndex];
        answer += filter[filterLength * filterLength * i + filterLength * j + k] * fieldValue;
      }
      base += fieldLength;
    }
    base -= filterLength * fieldLength;
    base += fieldLength * fieldLength;
  }
  result[gid] = answer;
}

extern "C"
__declspec(dllexport) void __cdecl
conv3d(float *field, float *filter, float *result, int fieldLength, int filterLength) {
  float *dField, *dFilter, *dResult;
  size_t fieldSize = sizeof(float) * fieldLength * fieldLength * fieldLength;
  size_t filterSize = sizeof(float) * filterLength * filterLength * filterLength;
  hipMalloc((void **) &dField, fieldSize);
  hipMalloc((void **) &dFilter, filterSize);
  hipMalloc((void **) &dResult, fieldSize);
  hipMemcpy(dField, field, fieldSize, hipMemcpyHostToDevice);
  hipMemcpy(dFilter, filter, filterSize, hipMemcpyHostToDevice);
  conv3d_gpu<<<fieldLength * fieldLength * fieldLength, 1>>>(dField, dFilter, dResult, fieldLength, filterLength);
  hipMemcpy(result, dResult, fieldSize, hipMemcpyDeviceToHost);
  hipFree(dFilter);
  hipFree(dResult);
  hipFree(dField);
}